#include <torch/extension.h>

torch::Tensor trilinear_fw_cu(torch::Tensor feats, torch::Tensor points){

    const int N = feats.size(0);
    const int F = feats.size(2);

    torch::Tensor feat_interp = torch::zeros({N, F}, feats.options());

    const dim3 threads(16, 16);
    const dim3 blocks((N + threads.x - 1) / threads.x, (F + threads.y - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_fw_cu",
    ([&] {
        trilinear_fw_kernel<scalar_t> <<<blocks, threads>>>(
            feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            feat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));


    return feats;
}